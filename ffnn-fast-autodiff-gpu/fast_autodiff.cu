#include "hip/hip_runtime.h"
#include <iostream>
#include <valarray>
#include <stdexcept>
#include "fast_autodiff.h"
#include <hipblas.h>
#include <hip/hip_runtime.h>

using namespace std;

namespace FA {


ostream& outputMatrix(ostream& os, float* m, unsigned int rows, unsigned int cols) {
    for(int row {0}; row< rows;row++) {
        for(int col {0}; col<cols;col++) {
            os << m[row*cols + col] << " ";
        }
        os << "\n";
    }
    
    return os;
}

void AD::compute(hipblasHandle_t *handle) {}

AD::AD(string _name, unsigned int _rows, unsigned int _cols)
    : name(_name)
    , rows(_rows)
    , cols(_cols) {
    hipMalloc((void**) &this->d_value, _rows * _cols * sizeof(float));
    hipMalloc((void**) &this->d_grad, _rows * _cols * sizeof(float));
    this->value = new float[_rows * _cols];
    this->resetGrad();

}

void AD::fromDevice() {
    hipError_t err;
    int size = this->rows * this->cols * sizeof(float);
    err = hipMemcpy(this->value, this->d_value, size, hipMemcpyDeviceToHost);
    if(err != hipSuccess) {
        printf("hipMemcpy failed at AD(%s)::fromDevice: %s\n", this->name, hipGetErrorName(err));
        exit(1);
    }
}

__global__ void doFill( int rows, int cols, float value, float* result) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int i = row * cols + col;
    if( i < rows*cols)
        result[i] = value;
}
void AD::computeGrad(hipblasHandle_t *handle) {
    float* seed;
    hipError_t err;
    int size = this->rows * this->cols * sizeof(float);
    err = hipMalloc((void**) &seed, size);
    if(err != hipSuccess) {
        printf("malloc error in Scalar::computeGrad: %s\n", hipGetErrorString(err));
        exit(1);
    }
    
    dim3 gd(ceil(this->cols/32.0), ceil(this->rows/32.0), 1);
    dim3 bd(32, 32, 1);
    doFill<<<gd, bd>>>( this->rows, this->cols, 1.0f, seed);


    
    this->pushGrad(handle, seed);
    
    err = hipDeviceSynchronize();
    if(err != hipSuccess) {
        printf("sync error in Scalar::computeGrad: %s\n", hipGetErrorString(err));
        exit(1);
    }

}

AD::~AD() {
    hipFree(this->d_grad);
    hipFree(this->d_value);
    delete [] this->value;
}


    
void AD::resetGrad() {
    hipMemset(this->d_grad, 0.0, this->rows * this->cols * sizeof(float));
}

void AD::pushGrad(hipblasHandle_t *handle, float* d_seed) {
    float alpha = 1;
    hipblasSaxpy(*handle, this->cols * this->rows, &alpha, d_seed, 1, this->d_grad, 1);
    hipFree(d_seed);
}

AbstractCol::AbstractCol(string _name, unsigned int _rows)
    : AD(_name, _rows, 1) {
}


void Col::loadValues(valarray<float> newValues) {
    if(newValues.size() != this->rows)
        throw out_of_range("size of col " + this->name + " (" + to_string(this->rows) + ") does not mathc size of valarray (" + to_string(newValues.size()) + ").");

    hipMemcpy(this->d_value, &(newValues[0]), this->rows * sizeof(float), hipMemcpyHostToDevice);

}

Col::Col(string _name, unsigned int _rows)
    : AbstractCol(_name, _rows){
}

__global__
void doGradDescent( float learningRate, int matrixCols, int matrixRows, float* matrix, float* grad) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if( (row < matrixRows) && (col < matrixCols)){
        int index = col + matrixCols * row;
        matrix[index] = matrix[index] - (learningRate * grad[index]);
    }


}
void Matrix::gradDescent(hipblasHandle_t *handle, float learningRate) {
    float alpha = 1;
    float beta = -1 * learningRate;

    hipblasSgeam(*handle,
                HIPBLAS_OP_N,
                HIPBLAS_OP_N,
                this->rows,
                this->cols,
                &alpha,
                this->d_value,
                this->rows,
                &beta,
                this->d_grad,
                this->rows,
                this->d_value,
                this->rows);
}

void Matrix::loadValues(valarray<float> newValues) {
    if(newValues.size() != this->rows * this->cols)
        throw out_of_range("size of matrix " + this->name + " (" + to_string(this->rows * this->cols) + ") does not match size of valarray (" + to_string(newValues.size()) + ").");
    
    int size = this->rows * this->cols * sizeof(float);

    hipError_t err = hipMemcpy(this->d_value, &(newValues[0]), size, hipMemcpyHostToDevice);
    if(err != hipSuccess) {
        printf("Matrix::loadValues unable to hipMemcpy: %s - %s\n", 
                hipGetErrorName(err),
                hipGetErrorString(err));
        exit(1);
    }


}


Matrix::Matrix(string _name, unsigned int _rows, unsigned int _cols)
    : AD(_name, _rows, _cols) {

}


void MatrixColProduct::resetGrad() {
    AD::resetGrad();
    this->matrix->resetGrad();
    this->col->resetGrad();

}



void MatrixColProduct::pushGrad(hipblasHandle_t *handle, float* d_seed) {
    // assert len(seed) == this->matrix->rows

    int matrixSize = this->matrix->rows * this->matrix->cols * sizeof(float);
    int colSize = this->col->rows * sizeof(float);
    float* matrixGrad;
    float* colGrad;

    hipMalloc((void**) &matrixGrad, matrixSize);
    hipMalloc((void**) &colGrad, colSize);

    float alpha = 1;
    float beta = 0;

    hipblasSgemm(*handle, 
                HIPBLAS_OP_T, 
                HIPBLAS_OP_N,
                this->matrix->cols,
                this->matrix->rows,
                1,  
                &alpha,
                this->col->d_value,
                1,  
                d_seed,
                1,  
                &beta,
                matrixGrad,
                this->matrix->cols);

    hipblasSgemm(*handle,
                HIPBLAS_OP_N,
                HIPBLAS_OP_T,
                1,
                this->matrix->cols,
                this->matrix->rows,
                &alpha,
                d_seed,
                1,
                this->matrix->d_value,
                this->matrix->cols,
                &beta,
                colGrad,
                1);


    this->matrix->pushGrad(handle, matrixGrad);
    this->col->pushGrad(handle, colGrad);
    hipFree(d_seed);


}


void MatrixColProduct::compute(hipblasHandle_t *handle) {
    this->matrix->compute(handle);
    this->col->compute(handle);

    float alpha = 1;
    float beta = 0;

    hipblasSgemv(*handle,
                HIPBLAS_OP_T,
                this->matrix->cols,
                this->matrix->rows,
                &alpha,
                this->matrix->d_value,
                this->matrix->cols,
                this->col->d_value,
                1,
                &beta,
                this->d_value,
                1);



}

MatrixColProduct::MatrixColProduct(AD* m, AbstractCol* x)
    : AbstractCol("Matrix product of " + m->name + " and " + x->name, m->rows)
    , matrix(m)
    , col(x) {

    if(m->cols != x->rows)
        throw invalid_argument("Input matrix " + m->name
                              + " has " + to_string(m->cols) + " rows but"
                              + " column vector has " + to_string(x->rows) 
                              + " columns.");

}

MatrixColProduct::~MatrixColProduct() {
    delete this->matrix;
    delete this->col;
}

void ColLeakyReLU::resetGrad() {
    AD::resetGrad();
    this->col->resetGrad();
}

__global__
void doComponentProduct( int rows
                       , float* grad
                       , float* seed
                       , float* result ) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    if( row < rows) {
        result[row] = seed[row] * grad[row];
    }

}

void ColLeakyReLU::pushGrad(hipblasHandle_t *handle, float* d_seed) {
    float* newSeed;
    hipError_t err;

    hipMalloc((void**) &newSeed, this->rows * sizeof(float));

    dim3 bd(1, 1024, 1);
    dim3 gd(1, ceil((this->col->rows)/1024.0), 1);

    doComponentProduct<<<gd, bd>>>(this->rows, this->d_grad, d_seed, newSeed);
    err = hipGetLastError();
    if(err != hipSuccess) {
        printf("Kernel launch error in ColLeakyReLU::pushGrad: %s\n", hipGetErrorString(err));
        exit(1);
    }
    hipDeviceSynchronize();

    this->col->pushGrad(handle, newSeed);
    hipFree(d_seed);
}


__global__
void doLeakyReLU( int Arows
                , int Acols
                , float* grad
                , float* A 
                , float* result) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if( (row < Arows) && (col < Acols)) {
        int i = row * Acols + col;
        if (A[i] > 0) {
            grad[i] = 1;
            result[i] = A[i];
        }else {
            result[i] = 0.01 * A[i];
            grad[i] = 0.01;
        }
    }
}

void ColLeakyReLU::compute(hipblasHandle_t *handle) {
    this->col->compute(handle);
    hipError_t err;

    dim3 bd (1, 1024, 1);
    dim3 gd (1, ceil((this->col->rows)/1024.0), 1);

    doLeakyReLU<<<gd, bd>>>( this->col->rows, 1, this->d_grad, this->col->d_value, this->d_value);
    err = hipGetLastError();
    if(err != hipSuccess) {
        printf("Kernel launch error in ColLeakyReLU::compute: %s\n", hipGetErrorString(err));
        exit(1);
    }

    hipDeviceSynchronize();


}

ColLeakyReLU::ColLeakyReLU(AbstractCol* _col)
    : col(_col)
    , AbstractCol( "ReLU of " + _col->name, _col->rows) {
}

ColLeakyReLU::~ColLeakyReLU() {
    delete this->col;
}

void Scalar::resetGrad() {
    AD::resetGrad();
    this->col->resetGrad();
}



void Scalar::pushGrad(hipblasHandle_t *handle, float* d_seed) {
    float *newSeed;
    hipMalloc((void**) &newSeed, this->col->rows * sizeof(float));
    hipblasScopy(*handle, this->col->rows, d_seed, 1, newSeed, 1);
    hipblasSscal(*handle, this->col->rows, &(this->scalar), newSeed, 1);

    hipFree(d_seed);
    this->col->pushGrad(handle, newSeed);
}


void Scalar::compute(hipblasHandle_t *handle) {
    this->col->compute(handle);

    hipblasScopy(*handle, this->col->rows, this->col->d_value, 1, this->d_value, 1);
    hipblasSscal(*handle, this->col->rows, &(this->scalar), this->d_value,1);

}

Scalar::Scalar(AbstractCol* _col, float _scalar)
    : col(_col)
    , scalar(_scalar)
    , AbstractCol( "Scalar of (" + _col->name + ") by "+ to_string(_scalar), _col->rows) {
}

Scalar::~Scalar() {
    delete this->col;
}


void AddCol::resetGrad() {
    AD::resetGrad();
    this->col1->resetGrad();
    this->col2->resetGrad();
}

void AddCol::pushGrad(hipblasHandle_t *handle, float* d_seed) {
    float* copySeed ;
    hipMalloc((void**) &copySeed, this->col1->rows * sizeof(float));
    hipMemcpy(copySeed, d_seed, this->col1->rows * sizeof(float), hipMemcpyDeviceToDevice);
    this->col1->pushGrad(handle, d_seed);
    this->col2->pushGrad(handle, copySeed);
}


void AddCol::compute(hipblasHandle_t *handle) {
    this->col1->compute(handle);
    this->col2->compute(handle);
    float alpha = 1;

    hipblasScopy(*handle, this->col1->rows, this->col1->d_value, 1, this->d_value, 1);

    hipblasSaxpy(*handle, this->col1->rows, &alpha, this->col2->d_value, 1, this->d_value, 1);



}

AddCol::AddCol(AbstractCol* _col1, AbstractCol* _col2) 
    : col1(_col1)
    , col2(_col2)
    , AbstractCol("Sum of (" + _col1->name + ") and (" + _col2->name + ")", _col1->rows) {
}

AddCol::~AddCol() {
    if( this->col1 == this->col2) {
        delete this->col1;
    } else {
        delete this->col1;
        delete this->col2;
    }
}

void InnerProduct::resetGrad() {
    AD::resetGrad();
    this->col1->resetGrad();
    this->col2->resetGrad();
}

void InnerProduct::pushGrad(hipblasHandle_t *handle, float* d_seed) {
    // assume len(seed)=1 here...
    float* vec1;
    float* vec2;
    float* scalar = new float;

    hipMemcpy(scalar, d_seed, sizeof(float), hipMemcpyDeviceToHost);

    hipMalloc((void**) &vec1, this->col1->rows * sizeof(float));

    hipMalloc((void**) &vec2, this->col2->rows * sizeof(float));


    hipblasScopy(*handle, this->col1->rows, this->col1->d_value, 1, vec1, 1);
    hipblasSscal(*handle, this->col1->rows, scalar, vec1,1);

    hipblasScopy(*handle, this->col2->rows, this->col2->d_value, 1, vec2, 1);
    hipblasSscal(*handle, this->col2->rows, scalar, vec2,1);

    
    this->col2->pushGrad(handle, vec1);
    this->col1->pushGrad(handle, vec2);

    delete scalar;

    hipFree(d_seed);
}

void InnerProduct::compute(hipblasHandle_t *handle) {
    this->col1->compute(handle);
    this->col2->compute(handle);

    hipblasSdot(*handle, this->col1->rows, this->col1->d_value, 1,
               this->col2->d_value, 1, this->d_value);

}

InnerProduct::InnerProduct(AbstractCol* _col1, AbstractCol* _col2) 
    : col1(_col1)
    , col2(_col2)
    , AbstractCol("Inner Product of (" + _col1->name + ") and (" + _col2->name + ")", 1) {
}

InnerProduct::~InnerProduct() {
    if( this->col1 == this->col2) {
        delete this->col1;
    } else {
        delete this->col1;
        delete this->col2;
    }
}

__global__ void doPadInput(float* input, float* paddedInput,
        int inputRows, int inputCols, int rowPadding, int colPadding) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int rows = inputRows + 2*rowPadding;
    int cols = inputCols + 2*colPadding;

    if(row < rows && col < cols) {
        if(row - rowPadding >= 0 && row < inputRows +rowPadding
            && col - colPadding >= 0 && col < inputCols + colPadding)
            paddedInput[row*cols + col] = input[(row - rowPadding) * inputCols + col - colPadding];
        else
            paddedInput[row*cols + col] = 0;
    }
}

void Convolution::padInput() {
    int paddedInputRows = this->multiplicand->rows + this->rowPadding * 2;
    int paddedInputCols = this->multiplicand->cols + this->colPadding * 2;

    hipError_t err; 
    dim3 gd(ceil(paddedInputCols/32.0), ceil(paddedInputRows/32.0), 1);
    dim3 bd(32, 32, 1);
    doPadInput<<<gd, bd>>>(this->multiplicand->d_value, this->d_input,
            this->multiplicand->rows, this->multiplicand->cols, 
            this->rowPadding, this->colPadding);
    err = hipGetLastError();
    if(err != hipSuccess) {
        printf("Kernel launch error in Convolution::padInput: %s\n", hipGetErrorString(err));
        exit(1);
    }

    hipDeviceSynchronize();




}


__global__ void doUnroll(float* kernel, float* matrix,
                         int kernelRows, int kernelCols,
                         int mRows, int mCols,
                         int inCols, int outCols,
                         int rowSkip, int colSkip) {
    int mrow = blockIdx.y * blockDim.y + threadIdx.y;
    int mcol = blockIdx.x * blockDim.x + threadIdx.x;
    if(mrow < mRows && mcol < mCols) {
        int outRow = mrow / outCols;
        int outCol = mrow % outCols;

        int inRow = mcol / inCols;
        int inCol = mcol % inCols;

        int kRowIndex = inRow - rowSkip * outRow;
        int kColIndex = inCol - colSkip * outCol;

        if(    kRowIndex >= 0 && kRowIndex < kernelRows 
            && kColIndex >=0 && kColIndex < kernelCols) {
            matrix[mrow*mCols + mcol] = kernel[kRowIndex * kernelCols + kColIndex];
        }else{
            matrix[mrow*mCols + mcol] = 0;
        }

    }
}

void Convolution::unrollKernel() {


    hipError_t err; 
    dim3 gd(ceil(this->unrKrnlCols/32.0), ceil(this->unrKrnlRows/32.0), 1);
    dim3 bd(32, 32, 1);
    doUnroll<<<gd, bd>>>(this->kernel->d_value, this->d_kernel,
             this->kernel->rows, this->kernel->cols,
             this->unrKrnlRows, this->unrKrnlCols,
             this->multiplicand->cols + 2*this->colPadding, this->cols,
             this->rowSkip, this->colSkip);
    err = hipGetLastError();
    if(err != hipSuccess) {
        printf("Kernel launch error in Convolution::unrollKerenl: %s\n", hipGetErrorString(err));
        exit(1);
    }

    hipDeviceSynchronize();
}

void Convolution::resetGrad() {
    AD::resetGrad();
    this->multiplicand->resetGrad();
    this->kernel->resetGrad();
}


// __global__ void doConvolution(int targetRows, int targetCols,
//                               int multiplicandRows, int multiplicandCols,
//                               int rowPadding, int rowSkip, int kernelRows,
//                               int colPadding, int colSkip, int kernelCols,
//                               float* multiplicand,
//                               float* kernel,
//                               float* result) {
//     int trow = blockIdx.y * blockDim.y + threadIdx.y;
//     int tcol = blockIdx.x * blockDim.x + threadIdx.x;
//     if(trow < targetRows && tcol < targetCols) {
//         float val = 0;
//         int mrow = - rowPadding + rowSkip * trow;
//         int mcol = - colPadding + colSkip * tcol;
// 
//         for(int i =mrow; i < mrow + kernelRows; i++)
//             for(int j =mcol; j< mcol + kernelCols; j++) 
//                 if(i >= 0 && j >= 0 && i < multiplicandRows && j < multiplicandCols) {
//                     int mIndex = multiplicandCols * i + j;
//                     int kIndex = kernelCols * (i-mrow) + (j-mcol);   
//                     val += multiplicand[mIndex] * kernel[kIndex];
//                 }
//         
// 
//         result[targetCols * trow + tcol] = val;
//     }
// }
// 
// 
// void Convolution::compute(hipblasHandle_t *handle) {
//     this->multiplicand->compute(handle);
//     this->kernel->compute(handle);
//     hipError_t err; 
//     dim3 gd(ceil(this->cols/32.0), ceil(this->rows/32.0), 1);
//     dim3 bd(32, 32, 1);
//     doConvolution<<<gd, bd>>>(this->rows, this->cols,
//                               this->multiplicand->rows, this->multiplicand->cols,
//                               this->rowPadding, this->rowSkip, this->kernel->rows,
//                               this->colPadding, this->colSkip, this->kernel->cols,
//                               this->multiplicand->d_value,
//                               this->kernel->d_value,
//                               this->d_value );
//     err = hipGetLastError();
//     if(err != hipSuccess) {
//         printf("Kernel launch error in Convolution::compute: %s\n", hipGetErrorString(err));
//         exit(1);
//     }
// 
//     hipDeviceSynchronize();
// }
//
__global__ void doKernelRoll(float* matrix, float* kernel,
                         int kernelRows, int kernelCols,
                         int mCols, int mRows,
                         int colSkip, int rowSkip,
                         int inCols, int outCols) {
    int mcol = blockIdx.x * blockDim.x + threadIdx.x;
    if(mcol < mCols) {
        int kRow = mcol / inCols;
        int kCol = mcol % inCols;
        if(    kRow >= 0 && kRow < kernelRows 
            && kCol >=0 && kCol < kernelCols) {
            float val = 0;
            for(int mrow = 0; mrow<mRows; mrow++) {
                int ocol = mrow % outCols;
                int orow = mrow / outCols;

                int offset = colSkip * ocol + rowSkip * orow * inCols;

                val += matrix[mrow*mCols + mcol + offset];

            }
            kernel[kRow * kernelCols + kCol] = val;
        }
    }
}

__global__ void doCopyWithoutPadding(float* paddedSource, float* dest,
        int rows, int cols, int rowPadding, int colPadding) {
     int row = blockIdx.y * blockDim.y + threadIdx.y;
     int col = blockIdx.x * blockDim.x + threadIdx.x;
     if(row < rows && col <cols) {
         dest[row*cols + col] = paddedSource[(row+ rowPadding)*(cols + 2* colPadding) + col + colPadding];
     }
}
                                    

void Convolution::pushGrad(hipblasHandle_t *handle, float* d_seed) {
    // assert len(seed) == this->matrix->rows

    int matrixSize = this->unrKrnlRows * this->unrKrnlCols * sizeof(float);
    int kernelSize = this->kernel->cols * this->kernel->rows * sizeof(float);
    int colSize = this->unrKrnlCols * sizeof(float);
    int inputSize = this->multiplicand->cols * this->multiplicand->rows * sizeof(float);
    float *rolledKernelGrad, *matrixGrad, *colGrad, *inputGrad;


    float alpha = 1;
    float beta = 0;

    hipMalloc((void**) &rolledKernelGrad, kernelSize);
    hipMalloc((void**) &matrixGrad, matrixSize);
    hipblasSgemm(*handle, 
                HIPBLAS_OP_T, 
                HIPBLAS_OP_N,
                this->unrKrnlCols,
                this->unrKrnlRows,
                1,  
                &alpha,
                this->d_input,
                1,  
                d_seed,
                1,  
                &beta,
                matrixGrad,
                this->unrKrnlCols);
    dim3 gd(ceil(this->unrKrnlCols/1024.0), 1, 1);
    dim3 bd(1024, 1, 1);
    doKernelRoll<<<gd, bd>>>(matrixGrad, rolledKernelGrad,
                  this->kernel->rows, this->kernel->cols,
                  unrKrnlCols, unrKrnlRows,
                  colSkip, rowSkip,
                  this->multiplicand->cols + 2*this->colPadding,
                  this->cols);
    hipFree(matrixGrad);
    this->kernel->pushGrad(handle, rolledKernelGrad);

    hipMalloc((void**) &colGrad, colSize);
    hipMalloc((void**) &inputGrad, inputSize);
    hipblasSgemm(*handle,
                HIPBLAS_OP_N,
                HIPBLAS_OP_T,
                1,
                this->unrKrnlCols,
                this->unrKrnlRows,
                &alpha,
                d_seed,
                1,
                this->d_kernel,
                this->unrKrnlCols,
                &beta,
                colGrad,
                1);
    dim3 gd2(ceil(this->multiplicand->cols/32.0), ceil(this->multiplicand->rows/32.0),1);
    dim3 bd2(32, 32, 1);
    doCopyWithoutPadding<<<gd2,bd2>>>(colGrad, inputGrad, this->multiplicand->rows, this->multiplicand->cols, this->rowPadding, this->colPadding);
    hipFree(colGrad);
    this->multiplicand->pushGrad(handle, inputGrad);



    hipFree(d_seed);


}

void Convolution::compute(hipblasHandle_t *handle) {
    this->multiplicand->compute(handle);
    this->kernel->compute(handle);
    this->padInput();
    this->unrollKernel();
    float alpha = 1;
    float beta = 0;

    hipblasSgemv(*handle,
                HIPBLAS_OP_T,
                this->unrKrnlCols,
                this->unrKrnlRows,
                &alpha,
                this->d_kernel,
                this->unrKrnlCols,
                this->d_input,
                1,
                &beta,
                this->d_value,
                1);

}


Convolution::Convolution(AD* m, AD* k,
        unsigned int rowPadding, unsigned int rowSkip,
        unsigned int colPadding, unsigned int colSkip)
    : multiplicand(m)
    , kernel(k)
    , rowPadding(rowPadding)
    , rowSkip(rowSkip)
    , colPadding(colPadding)
    , colSkip(colSkip)
    , AD("Convolution of "+m->name, (m->rows + 2 * rowPadding - k->rows)/rowSkip + 1, (m->cols + 2*colPadding - k->cols)/colSkip + 1){  

    this->unrKrnlCols = (this->multiplicand->rows + this->rowPadding *2) 
                             * (this->multiplicand->cols + this->colPadding * 2);
    this->unrKrnlRows = this->rows * this->cols;
    unsigned int unrKrnlSize = this->unrKrnlRows * this->unrKrnlCols;
    hipMalloc((void**) &this->d_kernel, unrKrnlSize * sizeof(float));
    hipMalloc((void**) &this->d_input, this->unrKrnlCols * sizeof(float));
}

Convolution::~Convolution() {
    delete this->multiplicand;
    delete this->kernel;
    hipFree(this->d_kernel);
    hipFree(this->d_input);
}

}
