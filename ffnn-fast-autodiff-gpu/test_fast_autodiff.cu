#include <iostream>
#include <valarray>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include "fast_autodiff.h"

using namespace std;
using namespace FA;

int main() {
    hipblasHandle_t cublasH;
    hipblasCreate(&cublasH);
    cout << "Testing Scalar \n";

    Col* xy = new Col("xy", 2);

    xy->loadValues({ 1.0, 2.0});
    
    Scalar* test_scalar = new Scalar(xy, 5);
    test_scalar->compute(&cublasH);
    test_scalar->fromDevice();

    if(  test_scalar->value[0] != 5.0
      || test_scalar->value[1] != 10.0 ) {
        cout << "Scalar failed!  should be {5, 10} but it is "
             << "{" << test_scalar->value[0] << ", " << test_scalar->value[1]
             << "}.\n";

    }

    cout << "Testing InnerProduct \n";

    Col* ab = new Col("ab", 2);
    ab->loadValues({ 3.0, 4.0 });
    InnerProduct* test_ip = new InnerProduct(xy, ab);
    test_ip->compute(&cublasH);
    test_ip->fromDevice();
    

    if( *test_ip->value != 11.0) {
        cout << "InnerProduct failed!  should be 11 but it is" << *test_ip->value << ".\n";
    }

    cout << "Testing AddCol \n";

    AddCol* test_add = new AddCol(xy, ab);
    test_add->compute(&cublasH);
    test_add->fromDevice();

    if (test_add->value[0] != 4 || test_add->value[1] != 6 ) {
        cout << "AddCol failed! Should be  {4, 6} but its" 
             << "{" << test_add->value[0] << ", " << test_add->value[1] << "}\n";
    }

    cout << "Testing MatrixColProduct\n";

    float *matrixGrad = new float[4];
    Matrix* abcd = new Matrix("abcd", 2, 2);
    abcd->loadValues({1,-1,-1, 1});
    hipMemcpy(matrixGrad, abcd->d_grad, 4 * sizeof(float), hipMemcpyDeviceToHost);

    cout << "Matrix grad\n"
         << matrixGrad[0] << ", " << matrixGrad[1] << "\n" 
         << matrixGrad[2] << ", " << matrixGrad[3] << "\n";

    MatrixColProduct *test_matCol = new MatrixColProduct(abcd, xy);
    test_matCol->compute(&cublasH);
    test_matCol->fromDevice();
    test_matCol->computeGrad(&cublasH);

    hipMemcpy(matrixGrad, abcd->d_grad, 4 * sizeof(float), hipMemcpyDeviceToHost);

    cout << "Matrix grad\n"
         << matrixGrad[0] << ", " << matrixGrad[1] << "\n" 
         << matrixGrad[2] << ", " << matrixGrad[3] << "\n";
    delete matrixGrad;

    xy->fromDevice();

    cout << "xy: {" << xy->value[0] << ", " << xy->value[1] << "}\n";


    if (test_matCol->value[0] != -1 || test_matCol->value[1] != 1 ) {
        cout << "MatrixColProduct failed! Should be  {-1, 1} but its" 
             << "{" << test_matCol->value[0] << ", " << test_matCol->value[1] << "}\n";
    }

    cout << "Testing Leaky ReLU\n";

    Col* z = new Col("z", 4);
    ColLeakyReLU* relu = new ColLeakyReLU(z);
    z->loadValues({500, -500, 0.5, -1});
    relu->compute(&cublasH);
    relu->fromDevice();

    cout << relu->value[0] << ", " << relu->value[1] << ", " << relu->value[2] << ", " << relu->value[3]
         << "\n";

    cout << "some Grad tests...\n";

    Col* x = new Col("x",1);
    InnerProduct* f = new InnerProduct(x, x);

    x->loadValues({3});
    f->compute(&cublasH);
    f->computeGrad(&cublasH);
    
    float* grad = new float;
    hipMemcpy(grad, x->d_grad, sizeof(float), hipMemcpyDeviceToHost);
    cout << "I'm expecting that d/dx (x^2) at x=3 is 6, but I computed: " << *grad << ".\n";
    f->resetGrad();

    x->loadValues({9});
    f->compute(&cublasH);
    f->computeGrad(&cublasH);
    
    hipMemcpy(grad, x->d_grad, sizeof(float), hipMemcpyDeviceToHost);
    cout << "I'm expecting that d/dx (x^2) at x=9 is 18, but I computed: " << *grad << ".\n";

    delete grad;
    
    cout << "Convolution tests...\n";
    Matrix* inputValues = new Matrix("input", 3, 3);
    Matrix* kernel = new Matrix("kernel", 2, 2);

    inputValues->loadValues({1,2,3,4,5,6,7,8,9});
    kernel->loadValues({3,3,3,3});


    Convolution* conv = new Convolution(inputValues, kernel, 0,1,0,1);

    conv->unrollKernel();

    float* testkernel = new float[conv->unrKrnlCols * conv->unrKrnlRows];
    hipMemcpy(testkernel, conv->d_kernel, sizeof(float)*conv->unrKrnlCols*conv->unrKrnlRows, hipMemcpyDeviceToHost);

    cout << "unrolled kernel looks like \n";
    outputMatrix(cout, testkernel, conv->unrKrnlRows, conv->unrKrnlCols);
    cout << "end unrolled kernel\n";


    conv->compute(&cublasH);

    float* testvalues = new float[4];
    hipMemcpy(testvalues, conv->d_value, sizeof(float)*4, hipMemcpyDeviceToHost);

    if(testvalues[0] != 36 || testvalues[1] != 48 || testvalues[2] != 72 || testvalues[3] != 84) {
        cout << "Convolution failed!  The output should be \n"
             << "( 36, 48 \n"
             << "  72, 84)\n"
             << "but it is\n";
        outputMatrix(cout, testvalues, 2, 2);
            
    }

    delete testvalues;
    delete testkernel;
    delete conv;

    inputValues = new Matrix("input", 4, 4);
    kernel = new Matrix("kernel", 3, 3);
    inputValues->loadValues({1,2,3,4,5,6,7,8,9,1,2,3,4,5,6,7});
    kernel->loadValues({1,0,0,0,1,0,0,0,1});


    conv = new Convolution(inputValues, kernel, 1, 3, 1, 3);

    conv->unrollKernel();

    testkernel = new float[conv->unrKrnlCols * conv->unrKrnlRows];
    hipMemcpy(testkernel, conv->d_kernel, sizeof(float)*conv->unrKrnlCols*conv->unrKrnlRows, hipMemcpyDeviceToHost);

    cout << "unrolled kernel looks like \n";
    outputMatrix(cout, testkernel, conv->unrKrnlRows, conv->unrKrnlCols);
    cout << "end unrolled kernel\n";

    conv->compute(&cublasH);
    testvalues = new float[4];
    hipMemcpy(testvalues, conv->d_value, sizeof(float)*4, hipMemcpyDeviceToHost);
    if(testvalues[0] != 7 || testvalues[1] != 4 || testvalues[2] != 4 || testvalues[3] != 9) {
        cout << "Convolution failed!  The output should be \n"
             << "( 7, 4 \n"
             << "  4, 9)\n"
             << "but it is\n";
        outputMatrix(cout, testvalues, 2, 2);
            
    }
    delete testvalues;
    delete testkernel;
    delete conv;






}
